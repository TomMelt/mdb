#include "hip/hip_runtime.h"
#include <stdio.h>
#include <mpi.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] - y[i];
}

int main(void)
{
  int N = 1<<10;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  int process_rank, size_of_cluster;

  MPI_Init(NULL, NULL);
  MPI_Comm_size(MPI_COMM_WORLD, &size_of_cluster);
  MPI_Comm_rank(MPI_COMM_WORLD, &process_rank);

  printf("process rank = %d\n", process_rank);

  if (process_rank >= 0){
    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    for (int j = 0; j < 2000; ++j) {
      for (int i = 0; i < N; i++) {
        x[i] = 1.0f * i * (process_rank + 1);
        y[i] = 2.0f * i * (process_rank + 1);
      }
      hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

      // Perform SAXPY on 1M elements
      saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

      hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
    }

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
      maxError = max(maxError, abs(y[i]));
    printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
  }
  free(x);
  free(y);
  MPI_Finalize();
}
